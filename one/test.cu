
#include <hip/hip_runtime.h>
class weekday {
  private:
    unsigned char __wd;
  public:
    weekday() = default;
    inline explicit constexpr weekday(unsigned __val) noexcept 
      : __wd(static_cast<unsigned char>(__val == 7 ? 0 : __val)) {}
    inline constexpr unsigned c_encoding()   const noexcept { return __wd;  }
};

constexpr int operator-(const weekday& __lhs, const weekday& __rhs) noexcept
{
  const int __wdu = __lhs.c_encoding() - __rhs.c_encoding();
  const int __wk = (__wdu >= 0 ? __wdu : __wdu-6) / 7;
  return __wdu - __wk * 7;
}

int main(void){
  constexpr weekday w0{0};
  constexpr weekday w6{6};
  static_assert((w0 - w6) == 1, "");
}
